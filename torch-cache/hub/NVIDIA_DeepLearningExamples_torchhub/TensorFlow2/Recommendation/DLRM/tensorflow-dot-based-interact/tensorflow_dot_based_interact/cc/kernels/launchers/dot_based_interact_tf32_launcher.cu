#include "hip/hip_runtime.h"
#ifndef TF32_LAUNCHER_CU
#define TF32_LAUNCHER_CU

#include "../cuda_kernels/dot_based_interact_tf32.cu"
#include "../launchers/dot_based_interact_fp32_launcher.cu"

inline void dotBasedInteractTF32Fwd(const void *input,
                                   const void *bottom_mlp_output,
                                   void *output,
                                   uint batch_size,
                                   uint num_rows,
                                   uint num_cols,
                                   hipStream_t stream) {
  const uint kWarpSize = 32;
  const uint kWarpSizeLog2 = Log2<kWarpSize>::value;
  const uint kTileLength = 16;
  const uint kTileLengthLog2 = Log2<kTileLength>::value;
  const uint kTileWidth = 8;
  const uint kTileWidthLog2 = Log2<kTileWidth>::value;
  const uint kWarpsPerBlock = 2;
  const uint kThreadBlockSize = kWarpsPerBlock * kWarpSize;
  const uint kRowTilesPerStep = 2;
  const uint kColTilesPerStep = 1;
  const uint kSkewFloat = 4;  // Ensures we are 16 byte align as required by nvcuda::wmma::load_matrix_sync

  // num tiles
  uint mat_a_num_row_tiles = (num_rows + kTileLength - 1) >> kTileLengthLog2;
  uint mat_a_num_col_tiles = (num_cols + kTileWidth - 1) >> kTileWidthLog2;

  // const uint &mat_b_num_row_tiles = mat_a_num_col_tiles;
  // const uint &mat_b_num_col_tiles = mat_a_num_row_tiles;

  // number of rows and columns after padding
  uint num_rows_after_padding = mat_a_num_row_tiles << kTileLengthLog2;
  uint num_cols_after_padding = mat_a_num_col_tiles << kTileWidthLog2;

  uint num_row_steps = mat_a_num_row_tiles / kRowTilesPerStep;
  uint num_col_steps = mat_a_num_col_tiles / kColTilesPerStep;

  const uint smem_stride = num_cols_after_padding + kSkewFloat;
  const uint smem_elems_per_warp_mat = num_rows_after_padding * smem_stride;

  const uint smem_stride_acc = num_rows_after_padding + kSkewFloat;
  const uint smem_elems_per_warp_acc = num_rows_after_padding * smem_stride_acc;

  const uint smem_elems_per_warp =
      smem_elems_per_warp_mat > smem_elems_per_warp_acc ? smem_elems_per_warp_mat : smem_elems_per_warp_acc;

  uint raw_output_size = num_cols + ((num_rows * (num_rows - 1)) >> 1);
  uint output_size = ((raw_output_size-1)/8 + 1)*8; //round up to multiple of 8
  uint padding_size = output_size-raw_output_size;
  bool float4_predicate = !((num_cols & 7) || (output_size & 7));

  if (float4_predicate) {
    dotBasedInteractTF32FwdKernel<kWarpsPerBlock,
                                  kThreadBlockSize,
                                  kWarpSize,
                                  kWarpSizeLog2,
                                  kTileLength,
                                  kTileLengthLog2,
                                  kTileWidth,
                                  kTileWidthLog2,
                                  kRowTilesPerStep>
        <<<(batch_size + kWarpsPerBlock - 1) / kWarpsPerBlock,
           kThreadBlockSize,
           kWarpsPerBlock * smem_elems_per_warp * sizeof(float), stream>>>((const float *)input,
                                                                           (float *)output,
                                                                           batch_size,
                                                                           num_rows,
                                                                           num_cols,
                                                                           num_rows_after_padding,
                                                                           num_cols_after_padding,
                                                                           smem_elems_per_warp,
                                                                           output_size,
                                                                           num_row_steps,
                                                                           num_col_steps,
                                                                           smem_stride,
                                                                           smem_stride_acc,
                                                                           padding_size);
  } else {
    // GENERIC VERSION IS UNFINISHED: Use FP32 instead for now
    dotBasedInteractFP32Fwd(input,
                                 bottom_mlp_output,
                                 output,
                                 batch_size,
                                 num_rows,
                                 num_cols,
                                 stream);
  }
}

inline void dotBasedInteractTF32Bwd(const void *input,
                                   const void *upstream_grad,
                                   void *grad,
                                   void *bottom_mlp_grad,
                                   uint batch_size,
                                   uint num_rows,
                                   uint num_cols,
                                   hipStream_t stream) {
  // Fragment Settings
  const uint kFragARows = 2;
  const uint kFragBCols = 2;
  const uint kTileLength = 16;
  const uint kTileLengthLog2 = Log2<kTileLength>::value;
  const uint kTileWidth = 8;
  const uint kTileWidthLog2 = Log2<kTileWidth>::value;

  const uint kWarpSize = 32;
  const uint kWarpSizeLog2 = Log2<kWarpSize>::value;
  const uint kSkewFloat = 4;
  const uint kWarpsPerBlock = 1;
  const uint kWarpsPerBlockLog2 = Log2<kWarpsPerBlock>::value;
  const uint kNumThreads = kWarpsPerBlock * kWarpSize;

  // num tiles
  uint mat_a_num_row_tiles = (num_rows + kTileLength - 1) >> kTileLengthLog2;
  uint mat_a_num_col_tiles = (num_rows + kTileWidth - 1) >> kTileWidthLog2;

  // const uint &mat_b_num_row_tiles = mat_a_num_col_tiles;
  uint mat_b_num_col_tiles = (num_cols + kTileLength - 1) >> kTileLengthLog2;

  // number of rows and columns after padding
  uint num_rows_after_padding = mat_a_num_row_tiles << kTileLengthLog2;
  uint num_cols_after_padding = mat_b_num_col_tiles << kTileLengthLog2;

  // 2D ugrad size and stride
  uint interaction_ugrad_2D_stride = num_rows_after_padding + kSkewFloat;
  uint interaction_ugrad_2D_size_elems = num_rows_after_padding * interaction_ugrad_2D_stride;

  // 1D ugrad size
  uint interaction_ugrad_size = num_rows * (num_rows - 1) >> 1;
  uint interaction_ugrad_size_with_padding = ((interaction_ugrad_size-1)/8 + 1)*8;

  // in_out place size and stride
  uint input_stride = num_cols_after_padding + kSkewFloat;
  uint input_size_elems = num_rows_after_padding * input_stride;

  // sample size
  uint sample_size = num_rows * num_cols;

  // output size
  uint output_size_elems = kTileLength * kTileLength * kFragARows * kFragBCols;

  // Shared memory size
  uint shared_mem_per_warp_size_elems = interaction_ugrad_2D_size_elems + input_size_elems + output_size_elems;
  uint shared_mem_size_elems = kWarpsPerBlock * shared_mem_per_warp_size_elems;
  uint shared_mem_size_bytes = shared_mem_size_elems * sizeof(float);

  uint num_blocks = (batch_size + kWarpsPerBlock - 1) >> kWarpsPerBlockLog2;
  uint num_k_steps = mat_a_num_col_tiles;
  uint num_n_steps = mat_b_num_col_tiles / kFragBCols;

  bool float4_predicate = !((interaction_ugrad_size_with_padding & 7) || (num_cols & 7));
  if (float4_predicate) {
    dotBasedInteractTF32BwdKernel<kWarpsPerBlock,
                                  kNumThreads,
                                  kWarpSize,
                                  kWarpSizeLog2,
                                  kFragARows,
                                  kFragBCols,
                                  kTileLength,
                                  kTileLengthLog2,
                                  kTileWidth,
                                  kTileWidthLog2>
        <<<num_blocks, kNumThreads, shared_mem_size_bytes, stream>>>((const float *)input,
                                                                     (const float *)upstream_grad,
                                                                     (float *)grad,
                                                                     (float *)bottom_mlp_grad,
                                                                     batch_size,
                                                                     num_rows,
                                                                     num_cols,
                                                                     num_rows_after_padding,
                                                                     num_cols_after_padding,
                                                                     sample_size,
                                                                     interaction_ugrad_size,
                                                                     interaction_ugrad_size_with_padding,
                                                                     interaction_ugrad_2D_size_elems,
                                                                     interaction_ugrad_2D_stride,
                                                                     input_size_elems,
                                                                     input_stride,
                                                                     shared_mem_per_warp_size_elems,
                                                                     num_k_steps,
                                                                     num_n_steps);
  } else {
    // GENERIC VERSION IS UNFINISHED: Use FP32 instead for now
    dotBasedInteractFP32Bwd(input,
                                 upstream_grad,
                                 grad,
                                 bottom_mlp_grad,
                                 batch_size,
                                 num_rows,
                                 num_cols,
                                 stream);
  }
}
#endif /* TF32_LAUNCHER_CU */